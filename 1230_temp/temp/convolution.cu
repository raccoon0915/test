#include "hip/hip_runtime.h"
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/version.hpp>
#include <opencv2/imgproc/imgproc.hpp>
using namespace cv;

#if CV_VERSION_EPOCH == 2
#define OPENCV2
#include <opencv2/gpu/gpu.hpp>
namespace GPU = cv::gpu;
#elif CV_VERSION_MAJOR == 4 
#define  OPENCV4
#include <opencv2/core/cuda.hpp>
namespace GPU = cv::cuda;
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/imgproc.hpp>
#include <iostream>

#define THREAD_X 32
#define THREAD_Y 32
#define WRAP_SIZE 32
#define MAX_WRAP_NUM 32

//using namespace cv;
//using namespace cv;

__constant__ double guass_kernel[2048];
__constant__ double guass_kernel_x[128*2];
__constant__ double guass_kernel_y[128];
int KERNEL_SIZE;

__global__ void convolution(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ int  share_mem[WRAP_SIZE][MAX_WRAP_NUM];
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
	//need to do bound check
	//printf("pixel %d %d block dim %d %d\n",pixel_i,pixel_j,blockDim.x,blockDim.y);
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
	/*int share_i=thread_block_index%WRAP_NUM;
	int share_j=thread_block_index/WRAP_NUM;*/
	double sum=0;
	//share_mem[share_i][share_j]=src(pixel_i,pixel_j);
	//share_mem[threadIdx.x][threadIdx.y]=src(pixel_i,pixel_j).x;
	//__syncthreads();
	 //printf("%d %d %d\n",pixel_i,pixel_j,share_mem[pixel_i][pixel_j]);
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
		int start_i=pixel_i-kernel_radius,start_j=pixel_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			for(int j=0;j<kernel_size;j++){
				int index_i=start_i+i,index_j=start_j+j;
				//sum+=share_mem[][index_j]*guass_kernel(i,j).x;
				sum+=src(index_j,index_i)*(float)guass_kernel[j];
			}
		}

		dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//sum;
	}
	return ;
}
//not need to padding
__global__ void conv_x(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[THREAD_Y+50][THREAD_X+50];
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;

	int shared_i=threadIdx.x+kernel_size/2;
	int shared_j=threadIdx.y;
	float sum=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	__syncthreads();
		int start_i=shared_i-kernel_radius,start_j=shared_j;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j][start_i+i]*(float)guass_kernel_x[i];
		}
		dst(pixel_j,pixel_i)=sum;//src(pixel_j,pixel_i);//sum;//sum;
	}
	return ;
}
__global__ void conv_y(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[THREAD_X+50][THREAD_Y+50];
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
        int shared_i=threadIdx.x;
	int shared_j=threadIdx.y+kernel_size/2;
	float sum=0;
//	for(int i=0;i<THREAD_X+20;i++)
//		for(int j=0;j<THREAD_X+20;j++)
//			share_mem[i][j]=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	
	__syncthreads();
		int start_i=shared_i,    start_j=shared_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j+i][start_i]*(float)guass_kernel_x[i];
			//sum+=share_mem[start_j+i][start_i];
		}
		dst(pixel_j,pixel_i)=sum;//share_mem[shared_j][shared_i];//sum;
	}
	return ;
}
void guassain_conv(const Mat *src,Mat *dst,double sigma){
//	int depth = CV_MAT_DEPTH(src.type());
	KERNEL_SIZE = cvRound(sigma* 4 * 2 + 1)|1;
	int kernel_radius=KERNEL_SIZE/2;
	int orign_width=src->cols,orign_height=src->rows;
	Mat padding_image;
	GPU::GpuMat device_image,g_kernel,result, dev_image,resul;

	if(GPU::getCudaEnabledDeviceCount()==0){
		std::cout<<"not use GPU module"<<std::endl;
		return ;
	}
	Mat gauss_x=getGaussianKernel(KERNEL_SIZE,sigma),gauss_y=getGaussianKernel(KERNEL_SIZE,sigma); //3*3 filter
	Mat gauss_kernel=gauss_x*gauss_y.t();
	//allocate
	double* gs_kernel,*dev_kernel;
	hipHostAlloc(&gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE,hipHostMallocDefault);
	for(int i=0;i<KERNEL_SIZE;i++){
		double* row=gauss_kernel.ptr<double>(i);
		for(int j=0;j<KERNEL_SIZE;j++){
			gs_kernel[i*KERNEL_SIZE+j]=row[j];
		}
	}
	hipMalloc(&dev_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	//allocate 
	//allocate
	double* x,*y;
	hipHostAlloc(&x,sizeof(double)*KERNEL_SIZE*2,hipHostMallocDefault);
	double *row_x=gauss_x.ptr<double>(0),*row_y=gauss_y.ptr<double>(0);
	for(int i=0;i<KERNEL_SIZE*2;i++){
		if(i<KERNEL_SIZE){
			x[i]=row_x[i];
			//std::cout<<x[i]<<std::endl;
		}
		else
			x[i]=row_y[i-KERNEL_SIZE];
	}
	//hipHostAlloc(&y,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	//allocate
	copyMakeBorder(*src,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	int grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	//int grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	result.upload(*dst);
	//g_kernel.upload(gauss_kernel);

	//use seperate do no padding
	//device_image.upload(padding_image);

	//device_image.upload(padding_image);
	device_image.upload(*src);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel_x),x,sizeof(double)*2*KERNEL_SIZE);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel),gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	dim3 thread_block(THREAD_X,THREAD_Y);
	dim3 grid(grid_num_x,grid_num_y);
	//convolution<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	conv_x<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	hipDeviceSynchronize();
	Mat re;
	result.download(re);
	resul.upload(re);
	//device_image.upload(*dst);
	conv_y<<<grid,thread_block>>>(resul,device_image,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	device_image.download(*dst);
	return ;
}



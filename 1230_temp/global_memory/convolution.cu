#include "hip/hip_runtime.h"
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/version.hpp>
#include <opencv2/imgproc/imgproc.hpp>
using namespace cv;

#if CV_VERSION_EPOCH == 2
#define OPENCV2
#include <opencv2/gpu/gpu.hpp>
namespace GPU = cv::gpu;
#elif CV_VERSION_MAJOR == 4 
#define  OPENCV4
#include <opencv2/core/cuda.hpp>
namespace GPU = cv::cuda;
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/imgproc.hpp>
#include <iostream>

#define THREAD_X 32
#define THREAD_Y 32
#define WRAP_NUM 32
#define MAX_WRAP_NUM 32

//using namespace cv;
//using namespace cv;

__constant__ double guass_kernel[2048];
__constant__ double guass_kernel_x[128*2];
__constant__ double guass_kernel_y[128];
int KERNEL_SIZE;

__global__ void convolution(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	//__shared__ int  share_mem[WRAP_SIZE][MAX_WRAP_NUM];
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
	//need to do bound check
	//printf("pixel %d %d block dim %d %d\n",pixel_i,pixel_j,blockDim.x,blockDim.y);
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
	/*int share_i=thread_block_index%WRAP_NUM;
	int share_j=thread_block_index/WRAP_NUM;*/
	double sum=0;
	//share_mem[share_i][share_j]=src(pixel_i,pixel_j);
	//share_mem[threadIdx.x][threadIdx.y]=src(pixel_i,pixel_j).x;
	//__syncthreads();
	 //printf("%d %d %d\n",pixel_i,pixel_j,share_mem[pixel_i][pixel_j]);
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
		int start_i=pixel_i-kernel_radius,start_j=pixel_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			for(int j=0;j<kernel_size;j++){
				int index_i=start_i+i,index_j=start_j+j;
				//sum+=share_mem[][index_j]*guass_kernel(i,j).x;
				sum+=src(index_j,index_i)*(float)guass_kernel[j];
			}
		}

		dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//sum;
	}
	return ;
}
//not need to padding
__global__ void conv_x(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/,GPU::PtrStepSz<double> gs_k,GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	/*int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;

	int shared_i=threadIdx.x+kernel_size/2;
	int shared_j=threadIdx.y;
	float sum=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	__syncthreads();
		int start_i=shared_i-kernel_radius,start_j=shared_j;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j][start_i+i]*(float)guass_kernel_x[i];
		}
		dst(pixel_j,pixel_i)=sum;//src(pixel_j,pixel_i);//sum;//sum;
	}*/
	int left_limit=kernel_radius,right_limit=blockDim.x-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x-2*blockIdx.x*kernel_radius;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
	//share_mem[threadIdx.y][threadIdx.x]=0;
	share_mem[thread_block_index%32][thread_block_index/32]=0;
	//share_mem[10]=src(pixel_j,pixel_i);
	__syncthreads();
	float sum=0,sum1=0,sum2=0;
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){//real image size
		share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
		__syncthreads();
		if(threadIdx.x>= left_limit && threadIdx.x<right_limit){ //non padding size
			int x=threadIdx.x-kernel_radius,y=threadIdx.y;

			for(int i=0;i<kernel_size;i++){
				
				thread_block_index=(x+i)+y*blockDim.x;
	//			if(thread_block_index>=2048 || thread_block_index<0)
	//				                                       printf("%d\n",thread_block_index/WRAP_NUM);
	//			if(x+i<100 && x+i>=0)
	//			sum+=share_mem[y][x+i]*(float)guass_kernel_x[i];
				//sum+=src(pixel_j,pixel_i-kernel_radius+i)*(float)guass_kernel_x[i];
				sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)gs_k[i];
				//sum1+=src(pixel_j,pixel_i-kernel_radius+i).y*(float)guass_kernel_x[i];
				//sum2+=src(pixel_j,pixel_i-kernel_radius+i).z*(float)guass_kernel_x[i];
			}
			dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//src(pixel_j,pixel_i);
			//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;
			//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;
		}
		//dst(pixel_j,pixel_i)=sum;	
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}
__global__ void conv_y(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<double> gs_k,GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	/*int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
        int shared_i=threadIdx.x;
	int shared_j=threadIdx.y+kernel_size/2;
	float sum=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	
	__syncthreads();
		int start_i=shared_i,    start_j=shared_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j+i][start_i]*(float)guass_kernel_x[i];
			//sum+=share_mem[start_j+i][start_i];
		}
		dst(pixel_j,pixel_i)=sum;//share_mem[shared_j][shared_i];//sum;
	}*/
	int top_limit=kernel_radius,down_limit=blockDim.y-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y-2*blockIdx.y*kernel_radius;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
//	share_mem[threadIdx.y][threadIdx.x]=0;
	share_mem[thread_block_index%32][thread_block_index/32]=0;
	__syncthreads();
	float sum=0.0,sum1=0,sum2=0;

	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
		share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
		__syncthreads();
		if(threadIdx.y>= top_limit && threadIdx.y<down_limit){
			int x=threadIdx.x,y=threadIdx.y-kernel_radius;
			for(int i=0;i<kernel_size;i++){
				thread_block_index=x+(y+i)*blockDim.x;
				sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)gs_k[i];
				//sum+=src(pixel_j-kernel_radius+i,pixel_i)*(float)guass_kernel_x[i];
//				if(y+i<100 && y>=0)
//				sum+=share_mem[y+i][x]*(float)guass_kernel_x[i];
				//sum1+=/*share_mem[thread_block_index%WRAP_NUM][thread_block_index/WRAP_NUM]src(pixel_j-kernel_radius+i,pixel_i).y*(float)guass_kernel_x[i];
				  //                              sum2+=src(pixel_j-kernel_radius+i,pixel_i).z*(float)guass_kernel_x[i];
			}
		
		dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//src(pixel_j,pixel_i);//sum;
//		dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;
//		                        dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;
		}
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}

void guassain_conv(const Mat *src,Mat *dst,double sigma){
//	int depth = CV_MAT_DEPTH(src.type());
	KERNEL_SIZE = cvRound(sigma* 4 * 2 + 1)|1;
	//std::cout<<KERNEL_SIZE<<std::endl;
	int kernel_radius=KERNEL_SIZE/2;
	int orign_width=src->cols,orign_height=src->rows;
	Mat padding_image;
	GPU::GpuMat device_image,g_kernel,result, dev_image,resul;

	if(GPU::getCudaEnabledDeviceCount()==0){
		std::cout<<"not use GPU module"<<std::endl;
		return ;
	}
	Mat gauss_x=getGaussianKernel(KERNEL_SIZE,sigma);//,gauss_y=getGaussianKernel(KERNEL_SIZE,sigma); //3*3 filter
	//Mat gauss_kernel=gauss_x*gauss_y.t();
	//allocate
	/*double* gs_kernel,*dev_kernel;
	hipHostAlloc(&gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE,hipHostMallocDefault);
	for(int i=0;i<KERNEL_SIZE;i++){
		double* row=gauss_kernel.ptr<double>(i);
		for(int j=0;j<KERNEL_SIZE;j++){
			gs_kernel[i*KERNEL_SIZE+j]=row[j];
		}
	}
	hipMalloc(&dev_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);*/
	//allocate 
	//allocate
	double* x,*y;
	hipHostAlloc(&x,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	double *row_x=gauss_x.ptr<double>(0);//,*row_y=gauss_y.ptr<double>(0);
	for(int i=0;i<KERNEL_SIZE;i++){
	//	if(i<KERNEL_SIZE){
			x[i]=row_x[i];
			//std::cout<<x[i]<<std::endl;
	//	}
	//	else
	//		x[i]=row_y[i-KERNEL_SIZE];
	}
	//hipHostAlloc(&y,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	//allocate
	copyMakeBorder(*src,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	int orign_grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,orign_grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	int grid_num_x=orign_grid_num_x+(2*kernel_radius*orign_grid_num_x+THREAD_X-1)/THREAD_X,grid_num_y=orign_grid_num_y+(2*kernel_radius*orign_grid_num_y+THREAD_Y-1)/THREAD_Y;
	//int grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	result.upload(*dst);
	g_kernel.upload(gauss_x);

	//use seperate do no padding
	//device_image.upload(padding_image);

	device_image.upload(padding_image);
	//device_image.upload(*src);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel_x),x,sizeof(double)*2*KERNEL_SIZE);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel),gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	dim3 thread_block(THREAD_X,THREAD_Y);
	dim3 grid(grid_num_x,grid_num_y);
	//convolution<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	conv_x<<<grid,thread_block>>>(device_image,g_kernel,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	hipDeviceSynchronize();
	Mat re;
	result.download(re);
	copyMakeBorder(re,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	//resul.upload(re);
	device_image.upload(padding_image);
	//result.upload(*dst);	
	conv_y<<<grid,thread_block>>>(device_image,g_kernel,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	result.download(*dst);
	return ;
}



#include "hip/hip_runtime.h"
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/version.hpp>
#include <opencv2/imgproc/imgproc.hpp>
using namespace cv;

#if CV_VERSION_EPOCH == 2
#define OPENCV2
#include <opencv2/gpu/gpu.hpp>
namespace GPU = cv::gpu;
#elif CV_VERSION_MAJOR == 4 
#define  OPENCV4
#include <opencv2/core/cuda.hpp>
namespace GPU = cv::cuda;
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/imgproc.hpp>
#include <iostream>

#define THREAD_X 32
#define THREAD_Y 32
#define WRAP_NUM 32
#define MAX_WRAP_NUM 32

//using namespace cv;
//using namespace cv;

__constant__ double guass_kernel_x[128*2];
__constant__ double guass_kernel_y[128];
int KERNEL_SIZE;

//not need to padding
__global__ void conv_x(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	/*int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;

	int shared_i=threadIdx.x+kernel_size/2;
	int shared_j=threadIdx.y;
	float sum=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	__syncthreads();
		int start_i=shared_i-kernel_radius,start_j=shared_j;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j][start_i+i]*(float)guass_kernel_x[i];
		}
		dst(pixel_j,pixel_i)=sum;//src(pixel_j,pixel_i);//sum;//sum;
	}*/
	int left_limit=kernel_radius,right_limit=blockDim.x-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x-2*blockIdx.x*kernel_radius;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
	//share_mem[threadIdx.y][threadIdx.x]=0;
	share_mem[thread_block_index%32][thread_block_index/32]=0;
	//share_mem[10]=src(pixel_j,pixel_i);
	__syncthreads();
	float sum=0,sum1=0,sum2=0;
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){//real image size
		share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
		__syncthreads();
		if(threadIdx.x>= left_limit && threadIdx.x<right_limit){ //non padding size
			int x=threadIdx.x-kernel_radius,y=threadIdx.y;

			for(int i=0;i<kernel_size;i++){
				
				thread_block_index=(x+i)+y*blockDim.x;
	//			if(thread_block_index>=2048 || thread_block_index<0)
	//				                                       printf("%d\n",thread_block_index/WRAP_NUM);
	//			if(x+i<100 && x+i>=0)
	//			sum+=share_mem[y][x+i]*(float)guass_kernel_x[i];
				//sum+=src(pixel_j,pixel_i-kernel_radius+i)*(float)guass_kernel_x[i];
				sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)guass_kernel_x[i];
				//sum1+=src(pixel_j,pixel_i-kernel_radius+i).y*(float)guass_kernel_x[i];
				//sum2+=src(pixel_j,pixel_i-kernel_radius+i).z*(float)guass_kernel_x[i];
			}
			dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//src(pixel_j,pixel_i);
			//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;
			//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;
		}
		//dst(pixel_j,pixel_i)=sum;	
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}
__global__ void conv_y(GPU::PtrStepSz<float> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<float> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	/*int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
        int shared_i=threadIdx.x;
	int shared_j=threadIdx.y+kernel_size/2;
	float sum=0;
	if(!(pixel_i>=orign_width || pixel_j>=orign_height)){
		share_mem[shared_j][shared_i]=src(pixel_j,pixel_i);
	
	__syncthreads();
		int start_i=shared_i,    start_j=shared_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			sum+=share_mem[start_j+i][start_i]*(float)guass_kernel_x[i];
			//sum+=share_mem[start_j+i][start_i];
		}
		dst(pixel_j,pixel_i)=sum;//share_mem[shared_j][shared_i];//sum;
	}*/
	int top_limit=kernel_radius,down_limit=blockDim.y-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y-2*blockIdx.y*kernel_radius;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
//	share_mem[threadIdx.y][threadIdx.x]=0;
	share_mem[thread_block_index%32][thread_block_index/32]=0;
	__syncthreads();
	float sum=0.0,sum1=0,sum2=0;

	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
		share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
		__syncthreads();
		if(threadIdx.y>= top_limit && threadIdx.y<down_limit){
			int x=threadIdx.x,y=threadIdx.y-kernel_radius;
			for(int i=0;i<kernel_size;i++){
				thread_block_index=x+(y+i)*blockDim.x;
				sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)guass_kernel_x[i];
				//sum+=src(pixel_j-kernel_radius+i,pixel_i)*(float)guass_kernel_x[i];
//				if(y+i<100 && y>=0)
//				sum+=share_mem[y+i][x]*(float)guass_kernel_x[i];
				//sum1+=/*share_mem[thread_block_index%WRAP_NUM][thread_block_index/WRAP_NUM]src(pixel_j-kernel_radius+i,pixel_i).y*(float)guass_kernel_x[i];
				  //                              sum2+=src(pixel_j-kernel_radius+i,pixel_i).z*(float)guass_kernel_x[i];
			}
		
		dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=sum;//src(pixel_j,pixel_i);//sum;
//		dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;
//		                        dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;
		}
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}

void guassain_conv(const Mat *src,Mat *dst,double sigma){
//	int depth = CV_MAT_DEPTH(src.type());
//	KERNEL_SIZE = cvRound(sigma* 4 * 2 + 1)|1;
	KERNEL_SIZE = 3;
//      std::cout<<KERNEL_SIZE<<std::endl;
	int kernel_radius=KERNEL_SIZE/2;
	int orign_width=src->cols,orign_height=src->rows;
	Mat padding_image;
	GPU::GpuMat device_image,g_kernel,result, dev_image,resul;

	if(GPU::getCudaEnabledDeviceCount()==0){
		std::cout<<"not use GPU module"<<std::endl;
		return ;
	}
	Mat gauss_x=getGaussianKernel(KERNEL_SIZE,sigma);//,gauss_y=getGaussianKernel(KERNEL_SIZE,sigma); //3*3 filter
//	Mat gauss_kernel=gauss_x*gauss_y.t();
	//allocate
/*	double* gs_kernel,*dev_kernel;
	hipHostAlloc(&gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE,hipHostMallocDefault);
	for(int i=0;i<KERNEL_SIZE;i++){
		double* row=gauss_kernel.ptr<double>(i);
		for(int j=0;j<KERNEL_SIZE;j++){
			gs_kernel[i*KERNEL_SIZE+j]=row[j];
		}
	}
	hipMalloc(&dev_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);*/
	//allocate 
	//allocate
	double* x,*y;
	hipHostAlloc(&x,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	double *row_x=gauss_x.ptr<double>(0);//,*row_y=gauss_y.ptr<double>(0);
	for(int i=0;i<KERNEL_SIZE;i++){
		//if(i<KERNEL_SIZE){
			x[i]=row_x[i];
			//std::cout<<x[i]<<std::endl;
		//}
		//else
		//	x[i]=row_y[i-KERNEL_SIZE];
	}
	//hipHostAlloc(&y,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	//allocate
	copyMakeBorder(*src,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	int orign_grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,orign_grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	int grid_num_x=orign_grid_num_x+(2*kernel_radius*orign_grid_num_x+THREAD_X-1)/THREAD_X,grid_num_y=orign_grid_num_y+(2*kernel_radius*orign_grid_num_y+THREAD_Y-1)/THREAD_Y;
	//int grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	result.upload(*dst);
	//g_kernel.upload(gauss_kernel);

	//use seperate do no padding
	//device_image.upload(padding_image);

	device_image.upload(padding_image);
	//device_image.upload(*src);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel_x),x,sizeof(double)*KERNEL_SIZE);
	//hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel),gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	dim3 thread_block(THREAD_X,THREAD_Y);
	dim3 grid(grid_num_x,grid_num_y);
	//convolution<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	conv_x<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	hipDeviceSynchronize();
	Mat re;
	result.download(re);
	copyMakeBorder(re,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	//resul.upload(re);
	device_image.upload(padding_image);
	//result.upload(*dst);	
	conv_y<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	result.download(*dst);
	return ;
}


